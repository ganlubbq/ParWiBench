#include "hip/hip_runtime.h"

#include <stdio.h>
#include "Scrambler.h"


void GenScrambInt(int *pScrambInt, int n)
{
	int i;
	int N_c = 1600;
	
	int n_init[31] = { 1, 1, 0, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 1, 1, 0, 1, 1, 1, 0};

	/////////////////////Generate ScrambSeq///////////////////////
	int px1[N_SCRAMB_IN_MAX + 1600];
	int px2[N_SCRAMB_IN_MAX + 1600];

	for (i = 0; i < 31; i++)
	{
		px1[i] = 0;
		px2[i] = n_init[i];
	}
	px1[0] = 1;
	
	for (i = 0; i < n + N_c - 31; i++)
	{
		px1[i + 31] =(px1[i + 3] + px1[i]) % 2;
		px2[i + 31] = (px2[i + 3] + px2[i + 2] + px2[i + 1] + px2[i]) % 2;
	}
	for (i = 0; i < n; i++)
	{
		pScrambInt[i] = 3; // What is this? Any use?
		pScrambInt[i] = (px1[i + N_c] + px2[i + N_c]) % 2;
	}
	/////////////////////END Generate ScrambSeq///////////////////////
}

__global__ void scramb_kernel(int *pInpSeq, int *scramb_seq_int, int *pOutSeq, int n_inp)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n_inp)
	{
		pOutSeq[i] = (pInpSeq[i] + scramb_seq_int[i]) % 2;
	}
}

void Scrambling(LTE_PHY_PARAMS *lte_phy_params, int *pInpSeq, int *pOutSeq)
{
	int n_inp;
//	int scramb_seq_int[N_SCRAMB_IN_MAX];
	int *scramb_seq_int;
	int *d_pInpSeq, *d_pOutSeq, *d_scramb_seq_int;

	int bdimx, gdimx;
//	int i;

	n_inp = lte_phy_params->scramb_in_buf_sz;
	bdimx = 32;
	gdimx = (n_inp + (bdimx - 1)) / bdimx;

	scramb_seq_int = (int *)malloc(n_inp * sizeof(int));

	hipMalloc((void **)&d_pInpSeq, n_inp * sizeof(int));
	hipMalloc((void **)&d_pOutSeq, n_inp * sizeof(int));
	hipMalloc((void **)&d_scramb_seq_int, n_inp * sizeof(int));
	
	GenScrambInt(scramb_seq_int, n_inp);

	hipMemcpy(d_pInpSeq, pInpSeq, n_inp * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_scramb_seq_int, scramb_seq_int, n_inp * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	scramb_kernel<<<gdimx, bdimx>>>(d_pInpSeq, d_scramb_seq_int, d_pOutSeq, n_inp);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("Elapsed time: %.3fms", time);

	//////////////////////// scrambling kernel ////////////////////////////
	/*
	for (i = 0; i < n_inp; i++)
	{
		pOutSeq[i] = (pInpSeq[i] + scramb_seq_int[i]) % 2;
	}
	*/
	////////////////////////END scrambling kernel ////////////////////////////

	hipMemcpy(pOutSeq, d_pOutSeq, n_inp * sizeof(int), hipMemcpyDeviceToHost);

	free(scramb_seq_int);

	hipFree(d_pInpSeq);
	hipFree(d_pOutSeq);
	hipFree(d_scramb_seq_int);
}


void Descrambling(LTE_PHY_PARAMS *lte_phy_params, float *pInpSeq, float *pOutSeq)
{
	int n_inp;
	float scramb_seq_float[N_SCRAMB_IN_MAX];
	int scramb_seq_int[N_SCRAMB_IN_MAX];

	int i;

	n_inp = lte_phy_params->scramb_in_buf_sz;
	// Generate integer scrambling sequence
	GenScrambInt(scramb_seq_int, n_inp);

	/*
	for (i = 0; i < n_inp; i++)
	{
		if (1 == scramb_seq_int[i])
		{
			scramb_seq_float[i] = -1.0;
		}
		else
		{
			scramb_seq_float[i] = +1.0;
		}
	}
	for (i = 0; i < n_inp; i++)
		pOutSeq[i] = pInpSeq[i] * scramb_seq_float[i];
	*/


	for (i = 0; i < n_inp; i++)
	{
		pOutSeq[i] = (pInpSeq[i] * (scramb_seq_int[i] * (-2.0) + 1.0));
	}
}
